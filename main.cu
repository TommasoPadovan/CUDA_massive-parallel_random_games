#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>
#include <host_defines.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
//#include "walker.h"
#include <iostream>
#include <fstream>

#define BLOCKS 2048
#define THREADS 1024
#define PER_THREAD_EXP 65536

__device__ int rng(hiprandState_t state) {
    return hiprand(&state) % 4;
}




//__device__ bool randomWalk(Walker a, Walker b, hiprandState_t state) {
//    a.setXY(0, 0);
//    b.setXY(2, 2);
//    for (int i=0; i<3; ++i) {
//        a.walk(state);
//        b.walk(state);
//        if (a.getX() == b.getX() && a.getY() == b.getY()) {
//            printf("sciabbe'\n");
//            return  true;
//        }
//    }
//    printf("me alone\n");
//    return false;
//}

__global__ void init(unsigned int seed, hiprandState_t* states) {
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &states[id]);
}

__global__ void simulate_kernel(hiprandState_t* states, int* C) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    int meetings = 0;
    for (int k = 0; k < PER_THREAD_EXP; ++k) {
        bool meet = false;
        int xa = 0;
        int ya = 0;
        int xb = 2;
        int yb = 2;
        for (int i = 0; i < 3; ++i) {
            int moveA = hiprand(&states[id]) % 4;
            int moveB = hiprand(&states[id]) % 4;
            switch (moveA) {
                case 0:
                    xa++;
                    break;
                case 1:
                    xa--;
                    break;
                case 2:
                    ya++;
                    break;
                case 3:
                    ya--;
                    break;
                default:
                    printf("really you shouldn't be here");
                    break;
            }
            switch (moveB) {
                case 0:
                    xb++;
                    break;
                case 1:
                    xb--;
                    break;
                case 2:
                    yb++;
                    break;
                case 3:
                    yb--;
                    break;
                default:
                    printf("really you shouldn't be here");
                    break;
            }
//            if (xa == xb && ya == yb) {
//                ++meetings;
//                break;
//            }
            //those lines remove the if to minimize warp divergence
            meet = meet || (xa == xb && ya == yb);
        }
        meetings += meet;
    }
//    printf("block %u, thread %u, meetings %u \n", blockIdx.x, threadIdx.x, meetings);
    atomicAdd(&C[blockIdx.x], meetings);
//    C[blockIdx.x] += meetings;

}



int main() {

    //init random states
    hiprandState_t* states;
    hipMalloc((void**) &states, BLOCKS * THREADS * sizeof(hiprandState_t));
    init<<<BLOCKS, THREADS>>>(time(0), states);

    //
    const unsigned int size = BLOCKS /* *THREADS */;
    int C[size] = {0};
    int *Cd = C;
    hipMalloc((void**) &Cd, size*sizeof(int));
    hipMemcpy(Cd, C, size*sizeof(int), hipMemcpyHostToDevice);

    simulate_kernel<<<BLOCKS,THREADS>>>(states, Cd);



    hipDeviceSynchronize();
    getLastCudaError("Kernel execution failed");

    //copying result back
    hipMemcpy(C, Cd, size*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(Cd);
    hipFree(states);
    

    long double success = 0;
    for (int i = 0; i<size; ++i) {
        success +=  (long double)C[i]/(long double)(THREADS * PER_THREAD_EXP);
    }
//    long long totExp = BLOCKS * THREADS * PER_THREAD_EXP;
//    std::cout << "P = " << std::scientific << success << " / " << totExp << std::endl;

    //hello
    std::ifstream swagFile("swag.txt");
    if (swagFile.is_open())
        std::cout << std::endl << swagFile.rdbuf() << std::endl << std::endl;
    printf("total experiments = %u * %u * %u \n", BLOCKS, THREADS, PER_THREAD_EXP);
//    printf("P = %llu / %llu \n", success, totExp);
    printf("P = %Le \n", success/(long double)BLOCKS);




    return 0;
}
