#include "hip/hip_runtime.h"
//
// Created by pado on 17/01/18.
//

#include <host_defines.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#define N 1000
#define THREADS 1000

__global__ void init(unsigned int seed, hiprandState_t* states) {
//    printf("-----------INIT--------------\n");
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
//    printf("id = %u \n", id);
    hiprand_init(seed, id, 0, &states[id]);
//    printf("hiprand init to %u, %u, %u\n", seed, id, 0);
//    printf("-----------INIT_END----------\n");

}

__global__ void randoms(hiprandState_t* states, unsigned int* numbers) {
//    printf("%u \n", blockDim.x);
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    numbers[id] = hiprand(&states[id]) % 100;
    for (int i=0; i<3; ++i) {
        printf("Block %u, Thread %u, id %u -> %u \n", blockIdx.x, threadIdx.x, id, hiprand(&states[id]) % 100);
    }
}


int main(void) {
    /* CUDA's random number library uses hiprandState_t to keep track of the seed value
     we will store a random state for every thread  */
    hiprandState_t* states;

    /* allocate space on the GPU for the random states */
    hipMalloc((void**) &states, N * THREADS * sizeof(hiprandState_t));

    /* invoke the GPU to initialize all of the random states */
    init<<<N, THREADS>>>(time(0), states);

    /* allocate an array of unsigned ints on the CPU and GPU */
    unsigned int cpu_nums[N];
    unsigned int* gpu_nums;
    hipMalloc((void**) &gpu_nums, N * sizeof(unsigned int));

    /* invoke the kernel to get some random numbers */
    randoms<<<N, THREADS>>>(states, gpu_nums);

    /* copy the random numbers back */
    hipMemcpy(cpu_nums, gpu_nums, N * sizeof(unsigned int), hipMemcpyDeviceToHost);

    /* print them out */
    for (int i = 0; i < N; i++) {
        printf("%u ", cpu_nums[i]);
    }
    printf("\n");

    /* free the memory we allocated for the states and numbers */
    hipFree(states);
    hipFree(gpu_nums);

    return 0;
}
